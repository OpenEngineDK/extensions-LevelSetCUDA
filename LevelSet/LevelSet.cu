#include "hip/hip_runtime.h"
// hello world

#include <Meta/CUDA.h>

#define GetPhi(phi,x,y,w) phi[x+w*(y)]

void cu_Init() {
    

}

__global__ void reinit(float *phi,float* phi0, float* phin, 
                       unsigned int width, unsigned int height) {
    uint x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    uint y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;

    if (x > width || y > height)
        return;
    
    float xy = GetPhi(phi,x,y,width);

    float phiXPlus = 0.0f;
    float phiXMinus = 0.0f;
    float phiYPlus = 0.0f;
    float phiYMinus = 0.0f;        	
    if (x != width-1) phiXPlus  = (GetPhi(phi,x+1, y,width) - xy);
    if (x != 0)       phiXMinus = (xy - GetPhi(phi,x-1, y,width));
    
    if (y !=height-1) phiYPlus  = (GetPhi(phi,x, y+1,width) - xy);
    if (y != 0)       phiYMinus = (xy - GetPhi(phi,x, y-1,width));

    /* GetPhi(phin,x,y,width) = phiYPlus; */
    /* return; */


    float dXSquared = 0;
    float dYSquared = 0;
    float a = GetPhi(phi0,x,y,width);
    if (a > 0) {
        // formula 6.3 page 58
        float _max = max(phiXMinus, 0.0f);
        float _min = min(phiXPlus, 0.0f);
        dXSquared = max(_max*_max, _min*_min);
                    
        _max = max(phiYMinus, 0.0f);
        _min = min(phiYPlus, 0.0f);
        dYSquared = max(_max*_max, _min*_min);
    } else {
        // formula 6.4 page 58
        float _max = max(phiXPlus, 0.0f);
        float _min = min(phiXMinus, 0.0f);
        dXSquared = max(_max*_max, _min*_min);
                    
        _max = max(phiYPlus, 0.0f);
        _min = min(phiYMinus, 0.0f);
        dYSquared = max(_max*_max, _min*_min);        				
    }

    float normSquared = dXSquared + dYSquared;           
    float norm = sqrt(normSquared);

    // Using the S(phi) sign formula 7.6 on page 67
    //float sign = phi(x,y) / sqrt(phi(x,y)*phi(x,y) + normSquared);
    float sign = GetPhi(phi0,x,y,width) / 
        sqrt(GetPhi(phi0,x,y,width)*GetPhi(phi0,x,y,width) + 1);
    float t = 0.3; // A stabil CFL condition
    GetPhi(phin,x,y,width) = GetPhi(phi,x,y,width) - sign*(norm - 1)*t;


}

void cu_Reinit(float* data, 
               unsigned int w,
               unsigned int h,
               unsigned int iterations) {
    float* phiData;
    float* phi0Data;
    float* phinData;
    /* int phiPitch; */
    /* int phi0Pitch; */
    /* int phinPitch; */

    /* hipArray* phiData; */
    /* hipArray* phi0Data; */
    /* hipArray* phinData; */

    /* hipChannelFormatDesc channelDesc = */
    /*     hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat); */
    

    /* hipMallocArray(&phiData,  &channelDesc, w, h); */
    /* hipMallocArray(&phi0Data, &channelDesc, w, h); */
    /* hipMallocArray(&phinData, &channelDesc, w, h); */

    /* hipMemcpyToArray(phiData,  0, 0, data, sizeof(float)*w*h, hipMemcpyHostToDevice); */
    /* hipMemcpyToArray(phi0Data, 0, 0, data, sizeof(float)*w*h, hipMemcpyHostToDevice); */
    /* hipMemcpyToArray(phinData, 0, 0, data, sizeof(float)*w*h, hipMemcpyHostToDevice); */


    hipMalloc((void**)&phiData, sizeof(float)*w*h);
    hipMalloc((void**)&phi0Data, sizeof(float)*w*h);
    hipMalloc((void**)&phinData, sizeof(float)*w*h);
    hipMemcpy((void*)phiData,(void*)data, sizeof(float)*w*h,hipMemcpyHostToDevice);
    hipMemcpy((void*)phi0Data,(void*)data, sizeof(float)*w*h,hipMemcpyHostToDevice);
    hipMemcpy((void*)phinData,(void*)data, sizeof(float)*w*h,hipMemcpyHostToDevice);

    /* hipMallocPitch((void**)&phiData, &phiPitch, sizeof(float)*w,h); */
    /* hipMallocPitch((void**)&phi0Data, &phi0Pitch, sizeof(float)*w,h); */
    /* hipMallocPitch((void**)&phinData, &phinPitch, sizeof(float)*w,h); */
    /* hipMemcpy((void*)phiData,(void*)data, sizeof(float)*w*h,hipMemcpyHostToDevice); */
    /* hipMemcpy((void*)phi0Data,(void*)data, sizeof(float)*w*h,hipMemcpyHostToDevice); */
    /* hipMemcpy((void*)phinData,(void*)data, sizeof(float)*w*h,hipMemcpyHostToDevice); */


    CHECK_FOR_CUDA_ERROR();

    const dim3 blockSize(32,16,1);
    const dim3 gridSize(w/blockSize.x, h/blockSize.y);

    //printf("%i,%i\n",w,h);

    //iterations=1;
    for (unsigned int i=0;i<iterations;i++) {
        reinit<<<gridSize,blockSize>>>(phiData,phi0Data,phinData,w,h);
        hipMemcpy((void*)phiData,(void*)phinData,sizeof(float)*w*h,hipMemcpyDeviceToDevice);
        hipDeviceSynchronize();
        CHECK_FOR_CUDA_ERROR();
    }

    hipMemcpy((void*)data,(void*)phiData, sizeof(float)*w*h,hipMemcpyDeviceToHost);
    CHECK_FOR_CUDA_ERROR();
    hipFree(phiData);
    hipFree(phi0Data);
    hipFree(phinData);


}